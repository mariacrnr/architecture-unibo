
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>




#define INF 1000000
#define BLKDIM 32


int* read_input(char* filename, int *source, int *V) {
    char folder[] = "input/";
    char* path = (char*) malloc(strlen(folder) + strlen(filename) + 1);
    strcpy(path, folder);

    FILE *file = fopen(strcat(path, filename), "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening file.\n");
        return NULL;
    }

    if (fscanf(file, "%d %d", V, source) != 2) {
        fprintf(stderr, "Error reading source and vertex count values.\n");
        fclose(file);
        return NULL;
    }

    int* graph = (int*) malloc((size_t)(*V) * (size_t)(*V) * sizeof(int *));
    if (graph == NULL) {
        fprintf(stderr, "Memory allocation error.\n");
        fclose(file);
        return NULL;
    }

    for (int i = 0; i < *V; i++) {
        for (int j = 0; j < *V; j++) {
            char token[10]; 
            if (fscanf(file, "%s", token) == 1) {
                if (strcmp(token, "INF") == 0) {
                    graph[i * (*V) + j] = INF;
                } else {
                    graph[i * (*V) + j] = atoi(token);
                }
            }
        }
    }

    fclose(file);

    return graph;
}

void print_output(char* filename, int V, int *distances, int has_negative){
    char folder[] = "output/cuda/";
    char* path = (char*) malloc(strlen(folder) + strlen(filename) + 1);
    strcpy(path, folder);

    FILE *file = fopen(strcat(path, filename), "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file.\n");
        return;
    }

    if(has_negative){
        fprintf(file, "Graph contains negative cycle!!");
    } else {
        for (int i = 0; i < V; i++) {
            fprintf(file, "%d\t\t%d\n", i, distances[i]);
        }
    }
    fprintf(file, "\n");

    fclose(file);
}

__global__ void bellmanford_kernel(int i, int V, int *graph, int *dist, int *has_changed, int *has_negative){
    int block_index = blockDim.x * blockIdx.x + threadIdx.x;
	int block_inc = blockDim.x * gridDim.x;

	if(block_index >= V) return;
	for(int u = 0 ; u < V ; u ++){
		for(int v = block_index; v < V; v+= block_inc){
            int updated_dist = graph[u * V + v] + dist[u];
            if (graph[u * V + v] < INF && updated_dist < dist[v]){
                dist[v] = updated_dist;
                *has_changed = 1;
            }
		}
	}

    __syncthreads();

    if(i == V-1 && has_changed) *has_negative = 1;

}

void bellmanford(int blocks_grid, int threads_block, int V, int *graph, int source, int *dist, int *has_negative, float *gpu_time){
    
    dim3 blocks(blocks_grid);
    dim3 threads(threads_block);

	int *d_graph, *d_dist;
	int *d_has_changed, h_has_changed;
    int *d_has_negative, h_has_negative;

	hipMalloc(&d_graph, sizeof(int) * V * V);
	hipMalloc(&d_dist, sizeof(int) * V);
	hipMalloc(&d_has_changed, sizeof(int));
    hipMalloc(&d_has_negative, sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time = 0;

	for(int i = 0 ; i < V; i ++){
		dist[i] = INF;
	}
	dist[source] = 0;

	hipMemcpy(d_graph, graph, sizeof(int) * V * V, hipMemcpyHostToDevice);
	hipMemcpy(d_dist, dist, sizeof(int) * V, hipMemcpyHostToDevice);

    h_has_negative = 0;

	for(int i = 0; i < V; i++){
		h_has_changed = 0;
		hipMemcpy(d_has_changed, &h_has_changed, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_has_negative, &h_has_negative, sizeof(int), hipMemcpyHostToDevice);

        hipEventRecord(start);
		bellmanford_kernel<<<blocks, threads>>>(i, V, d_graph, d_dist, d_has_changed, d_has_negative);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        *gpu_time += time;

		hipMemcpy(&h_has_changed, d_has_changed, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(has_negative, d_has_negative, sizeof(int), hipMemcpyHostToHost);

		if(!h_has_changed || h_has_negative){
			break;
		}

	}

	if(! *has_negative){
		hipMemcpy(dist, d_dist, sizeof(int) * V, hipMemcpyDeviceToHost);
	}


    printf("%d", *has_negative);

	hipFree(d_graph);
	hipFree(d_dist);
	hipFree(d_has_changed);
}

int main(){

    char filename[] = "simple.txt";

    int source, V, has_negative;
    int *graph = read_input(filename, &source, &V);
    if(graph == NULL) return 1;

    int *dist = (int*) malloc(sizeof(int) * (size_t)V);

    float gpu_time = 0;

    hipDeviceReset();
    bellmanford(1, 1, V, graph, source, dist, &has_negative, &gpu_time);
    hipDeviceSynchronize();

    printf("Elapsed Time: %f milliseconds\n", gpu_time);

    print_output(filename, V, dist, has_negative);

    free(dist);
    free(graph);

    return 0;
}